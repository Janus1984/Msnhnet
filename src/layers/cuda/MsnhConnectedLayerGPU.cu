#include "hip/hip_runtime.h"
﻿#include "Msnhnet/layers/cuda/MsnhConnectedLayerGPU.h"

namespace Msnhnet
{

__global__ void connBnKernel(const int n, const int outChannel, const int outWxH, float *const gpuScales,
                             float *const gpuRollMean, float *const gpuRollVariance, float *const gpuBiases, const float &eps, float *const gpuOutput)
{
    int index   = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

    if(index < n)
    {
        int i = index % outWxH;
        index = index / outWxH;
        int c = index % outChannel;
        index = index / outChannel;
        int b = index;

        int id = b*outChannel*outWxH + c*outWxH + i;

        gpuOutput[id]  = gpuScales[c]*(gpuOutput[id] - gpuRollMean[c])/sqrtf(gpuRollVariance[c] + eps) + gpuBiases[c];
    }
}

void ConnectedLayerGPU::connBn(const int &batch, const int &outChannel, const int &outHeight, const int &outWidth, float* const &gpuScales,
                                     float *const &gpuRollMean, float *const &gpuRollVariance, float *const &gpuBiases, const float &eps, float *const &gpuOutput)
{
    int num     = batch*outChannel*outWidth*outHeight;
    int outWxH  = outHeight*outWidth;

    connBnKernel<<<Cuda::getGrid(num), Cuda::blockThread, 0, Cuda::getCudaStream()>>>(num, outChannel, outWxH, gpuScales,
                                                                                      gpuRollMean, gpuRollVariance, gpuBiases, eps, gpuOutput);

    CUDA_CHECK(hipPeekAtLastError());
}
}
