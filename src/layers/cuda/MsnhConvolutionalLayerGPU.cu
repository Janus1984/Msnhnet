#include "hip/hip_runtime.h"
﻿#include "Msnhnet/layers/cuda/MsnhConvolutionalLayerGPU.h"

namespace Msnhnet
{

__global__ void convBnKernel(const int n, const int outChannel, const int outWxH, float *const gpuScales,
                             float *const gpuRollMean, float *const gpuRollVariance, float *const gpuBiases, const float bnEps, float *const gpuOutput)
{
    int index   = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

    if(index < n)
    {
        int i = index % outWxH;
        index = index / outWxH;
        int c = index % outChannel;
        index = index / outChannel;
        int b = index;

        int id = b*outChannel*outWxH + c*outWxH + i;

        gpuOutput[id]  = gpuScales[c]*(gpuOutput[id] - gpuRollMean[c])/sqrtf(gpuRollVariance[c] + bnEps) + gpuBiases[c];
    }
}

void ConvolutionalLayerGPU::convBn(const int &batch, const int &outChannel, const int &outHeight, const int &outWidth, float* const &gpuScales,
                                     float *const &gpuRollMean, float *const &gpuRollVariance, float *const &gpuBiases, const float &bnEps, float *const &gpuOutput)
{
    int num     = batch*outChannel*outWidth*outHeight;
    int outWxH  = outHeight*outWidth;

    convBnKernel<<<Cuda::getGrid(num), Cuda::blockThread, 0, Cuda::getCudaStream()>>>(num, outChannel, outWxH, gpuScales,
                                                                                      gpuRollMean, gpuRollVariance, gpuBiases, bnEps, gpuOutput);

    CUDA_CHECK(hipPeekAtLastError());
}
}
